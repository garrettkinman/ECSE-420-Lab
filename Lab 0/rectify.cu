#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void rectifyParallel(unsigned char* og_img, unsigned char* new_img, unsigned int num_thread, unsigned int size)
{
    // iterate through all the blocks, same threadIdx for each
    for (int i = threadIdx.x; i < size; i += num_thread) {
        if (og_img[i] < 127) {
            new_img[i] = 127;
        }
        else {
            new_img[i] = og_img[i];
        }
    }
}

void rectifySequential(unsigned char* og_img, unsigned char* new_img, unsigned int num_thread, unsigned int size) {
    // iterate through all elements of og_img
    for (int i = 0; i < sizeof(og_img) / sizeof(og_img[0]); i++) {
        if (og_img[i] < 127) {
            new_img[i] = 127;
        }
        else {
            new_img[i] = og_img[i];
        }
    }
}

int main(int argc, char *argv[]) {
    // TODO:
    // 1) read in and validate arguments
    // 2) load in input png from file
    // 3) make variables available to both CPU and GPU
    // 4) specify launch config of kernel function
    // 5) call parallelized rectify function, record performance
    // 6) call sequential rectify function, record performance
    // 7) write output image from parallelized rectify function to file

    return 0;
}