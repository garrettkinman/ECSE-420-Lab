#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "lodepng.h"

#include <stdio.h>

__global__ void rectifyParallel(unsigned char* original_img, unsigned char* new_img, unsigned int num_threads, unsigned int img_size)
{
    // iterate through all the blocks, same threadIdx for each
    for (int i = threadIdx.x; i < img_size; i += num_threads) {
        if (original_img[i] < 127) {
            new_img[i] = 127;
        }
        else {
            new_img[i] = original_img[i];
        }
    }
}

void rectifySequential(unsigned char* original_img, unsigned char* new_img, unsigned int img_size) {
    // iterate through all elements of og_img
    for (int i = 0; i < img_size; i++) {
        if (original_img[i] < 127) {
            new_img[i] = 127;
        }
        else {
            new_img[i] = original_img[i];
        }
    }
}

int main(int argc, char *argv[]) {

    // ~~~~~~~~~~~~~~~~~~~~~~~
    // step 1: parse arguments
    // ~~~~~~~~~~~~~~~~~~~~~~~

    if (argc != 4) {
        printf("Error: Input arguments are of format:\n./rectify <input filename> <output filename> <# threads>");
        return -1;
    }

    int input_filename_len = strlen(argv[1]);
    int output_filename_len = strlen(argv[2]);

    // dynamically allocate strings of appropriate size to hold filenames
    char *input_filename = (char*)malloc(input_filename_len * sizeof(char));
    char *output_filename = (char*)malloc(output_filename_len * sizeof(char));

    strcpy(input_filename, argv[1]);
    strcpy(output_filename, argv[2]);

    unsigned int num_threads = atoi(argv[3]);

    if (num_threads < 1) {
        printf("Error: '%u' is an invalid number of threads.\nNumber of threads must be greater than zero.", num_threads);
        return -1;
    }

    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // step 2: read in input image from file
    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

    unsigned char* original_img;
    unsigned char* new_img;
    unsigned int img_width, img_height;

    int error = lodepng_decode32_file(&original_img, &img_width, &img_height, input_filename);
    if (error) {
        printf("Error %d: %s\n", error, lodepng_error_text(error));
        return -1;
    }

    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // step 3: make variables available to both CPU and GPU
    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

    unsigned char* original_img_cuda;
    unsigned char* new_img_cuda;

    // allocate for CPU
    unsigned int img_size = img_width * img_height * 4 * sizeof(unsigned char);
    new_img = (unsigned char*)malloc(img_size);

    // allocate for GPU
    hipMalloc((void**)&original_img_cuda, img_size);
    hipMalloc((void**)&new_img_cuda, img_size);
    hipMemcpy(original_img_cuda, original_img, img_size, hipMemcpyHostToDevice);

    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // step 4: call parallelized rectify function, record performance
    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

    // TODO: measure time
    rectifyParallel<<<1, num_threads>>>(original_img_cuda, new_img_cuda, num_threads, img_size);

    hipDeviceSynchronize();
    hipMemcpy(new_img, new_img_cuda, img_size, hipMemcpyDeviceToHost);

    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // step 5: write output image from parallelized rectify function to file
    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

    error = lodepng_encode32_file(output_filename, new_img, img_width, img_height);
    if (error) {
        printf("Error %d: %s\n", error, lodepng_error_text(error));
        return -1;
    }

    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // step 6: call sequential rectify function, record performance
    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

    // TODO: measure time
    rectifySequential(original_img, new_img, img_size);

    // ~~~~~~~~~~~~~~~~~~~~~
    // step 7: free at last!
    // ~~~~~~~~~~~~~~~~~~~~~

    /*
    free(input_filename);
    free(output_filename);
    free(original_img);
    free(new_img);
    hipFree(original_img_cuda);
    hipFree(new_img_cuda);
    */

    return 0;
}