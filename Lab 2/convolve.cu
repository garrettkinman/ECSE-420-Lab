#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "lodepng.h"
#include "wm.h"

#include <stdio.h>
#include <time.h>
#include <math.h>

typedef struct Image {
    unsigned char* img;
    unsigned int size;
    unsigned int width;
    unsigned int height;
} Image;

unsigned char clampedWeightedSum(unsigned char* img, float* weights, unsigned width, unsigned start, unsigned color) {
    // perform convolution
    float sum = 0.0;
    for (int i = 0; i < 9; i++) {
        int index = start + (i / 3) * width + (i % 3);
        sum += (int)img[4 * index + color] * weights[i];
    }

    // clamp to fit as unsigned char
    if (sum > 255) {
        sum = 255;
    }
    if (sum < 0) {
        sum = 0;
    }

    // round and cast
    return (unsigned char)roundf(sum);
}

__device__ unsigned char clampedWeightedSumKernel(unsigned char* img, float* weights, unsigned width, unsigned start, unsigned color) {
    // perform convolution
    float sum = 0.0;
    for (int i = 0; i < 9; i++) {
        int index = start + (i / 3) * width + (i % 3);
        sum += (int) img[4 * index + color] * weights[i];
    }

    // clamp to fit as unsigned char
    if (sum > 255) {
        sum = 255;
    }
    if (sum < 0) {
        sum = 0;
    }

    // round and cast
    return (unsigned char)sum;
}

__global__ void convolveParallel(unsigned char* original, unsigned char* convolved, float* weights, unsigned width, unsigned height, int n_threads, unsigned windows_per_thread) {
    int windows_done_so_far = windows_per_thread * threadIdx.x;

    for (int i = 0; i < windows_per_thread; i++) {
        int window_start = ((windows_done_so_far + i) / (width - 2)) * (width)+((windows_done_so_far + i) % (width - 2));
        for (int color = 0; color < 4; color++) {
            float result = 0.0;
            if (color == 3) {
                result = 255;
            }
            else {
                result = clampedWeightedSumKernel(original, weights, width, window_start, color);
            }
            convolved[(windows_done_so_far + i) * 4 + color] = result;
        }
    }
}

//void convolveSequential(Image* original, Image* convolved) {
//    // increment by 4 values, as each pixel has 4 channels: RGBA
//    int increment = 4 * sizeof(unsigned char);
//    // k is index in the output image
//    int k = 0;
//    // width is in bytes, original->width is in pixels
//    int height = 4 * original->width * sizeof(unsigned char);
//    for (int i = 0; i < original->size; i += increment) {
//        // if outermost pixel of original image, correct k
//        // where image is row-major (TODO: make sure it is actually row-major and not column-major)
//        if (i % height == 0 || (i + increment) % height == 0 || i - height < 0 || i + height > original->size) {
//            k -= increment;
//        }
//        else {
//            // else, convolve with weight matrix (don't need to convolve alpha channel)
//            for (int j = 0; j < 3 * sizeof(unsigned char); j += sizeof(unsigned char)) {
//                convolved->img[k + j] = clampedWeightedSum(weights, original, i, j);
//            }
//        }
//
//    }
//}

int main(int argc, char* argv[]) {

    // ~~~~~~~~~~~~~~~~~~~~~~~
    // step 1: parse arguments
    // ~~~~~~~~~~~~~~~~~~~~~~~

    if (argc != 4) {
        printf("Error: Input arguments are of format:\n./convolve <input filename> <output filename> <# threads>");
        return -1;
    }

    unsigned int n_threads = atoi(argv[3]);

    if (n_threads < 1) {
        printf("Error: '%u' is an invalid number of threads.\nNumber of threads must be greater than zero.", n_threads);
        return -1;
    }

    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // step 2: read in input image from file
    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

    //Image original;
    //Image convolved;
    unsigned char* original;
    unsigned char* convolved;
    unsigned width, height;

    int error = lodepng_decode32_file(&original, &width, &height, argv[1]);
    if (error) {
        printf("Error %d: %s\n", error, lodepng_error_text(error));
        return -1;
    }

    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // step 3: allocate for CPU and GPU
    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

    //Image original_cuda;
    //Image convolved_cuda;

    // allocate for CPU
    //original.size = original.width * original.height * 4 * sizeof(unsigned char);
    //convolved.size = (original.width - 2) * (original.height - 2) * 4 * sizeof(unsigned char);
    //convolved.img = (unsigned char*)malloc(convolved.size);

    // allocate for CPU
    size_t original_size = width * height * 4 * sizeof(unsigned char);
    size_t convolved_size = (width - 2) * (height - 2) * 4 * sizeof(unsigned char);
    convolved = (unsigned char*)malloc(convolved_size);
    
    // allocate for GPU
    unsigned char* original_cuda;
    unsigned char* convolved_cuda;
    float* w_cuda;

    hipMalloc((void**)&original_cuda, original_size);
    hipMalloc((void**)&convolved_cuda, convolved_size);
    hipMalloc((void**)&w_cuda, 3 * 3 * sizeof(float));

    hipMemcpy(original_cuda, original, original_size, hipMemcpyHostToDevice);
    hipMemcpy(convolved_cuda, convolved, convolved_size, hipMemcpyHostToDevice);
    hipMemcpy(w_cuda, w[0], 3 * 3 * sizeof(float), hipMemcpyHostToDevice);


    // allocate for GPU
    //hipMalloc((void**)&(original_cuda.img), original.size);
    //hipMalloc((void**)&(convolved_cuda.img), convolved.size);
    //hipMalloc((void**)&original_cuda, sizeof(original));
    //hipMalloc((void**)&convolved_cuda, sizeof(convolved));
    //hipMemcpy(&original_cuda, &original, original.size, hipMemcpyHostToDevice);
    //hipMemcpy(original_cuda.img, original.img, original.size, hipMemcpyHostToDevice);

    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // step 6: convolve sequentially, record performance
    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

    //convolveSequential(&original, &convolved);
    //error = lodepng_encode32_file(argv[2], convolved.img, convolved.width, convolved.height);

    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // step 7: convolve in parallel, record performance
    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

    unsigned windows_per_thread = ((width - 2) * (height - 2)) / n_threads;
    convolveParallel<<<1, n_threads>>>(original_cuda, convolved_cuda, w_cuda, width, height, n_threads, windows_per_thread);

    hipMemcpy(convolved, convolved_cuda, convolved_size, hipMemcpyDeviceToHost);
    
    error = lodepng_encode32_file(argv[2], convolved, width - 2, height - 2);
    if (error) {
        printf("Error %d: %s\n", error, lodepng_error_text(error));
        return -1;
    }

    // ~~~~~~~~~~~~~~~~~~~~~
    // step 9: free at last!
    // ~~~~~~~~~~~~~~~~~~~~~

    free(original);
    free(convolved);
    hipFree(original_cuda);
    hipFree(convolved_cuda);
    hipFree(w_cuda);
}