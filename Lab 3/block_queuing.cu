#ifndef CUDACC
#define CUDACC
#endif

#include "hip/hip_runtime.h"


#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define AND 0
#define OR 1
#define NAND 2
#define NOR 3
#define XOR 4
#define NXOR 5

inline hipError_t checkCudaErr(hipError_t err, const char* msg) {
  if (err != hipSuccess)
      fprintf(stderr, "CUDA Runtime error at %s: %s\n", msg, hipGetErrorString(err));
  return err;
}

int read_input_one_two_four(int **input1, char* filepath) {
    FILE* fp = fopen(filepath, "r");
    if (fp == NULL){
        fprintf(stderr, "Couldn't open file for reading\n");
        exit(1);
    } 
  
    int counter = 0;
    int len;
    int length = fscanf(fp, "%d", &len);
    *input1 = ( int *)malloc(len * sizeof(int));

    int temp1;

    while (fscanf(fp, "%d", &temp1) == 1) {
        (*input1)[counter] = temp1;
        counter++;
    }

    fclose(fp);
    return len;
}

int read_input_three(int** input1, int** input2, int** input3, int** input4, char* filepath){
    FILE* fp = fopen(filepath, "r");
    if (fp == NULL){
        fprintf(stderr, "Couldn't open file for reading\n");
        exit(1);
    } 
  
    int counter = 0;
    int len;
    int length = fscanf(fp, "%d", &len);
    *input1 = ( int *)malloc(len * sizeof(int));
    *input2 = ( int *)malloc(len * sizeof(int));
    *input3 = ( int *)malloc(len * sizeof(int));
    *input4 = ( int *)malloc(len * sizeof(int));

    int temp1;
    int temp2;
    int temp3;
    int temp4;
    while (fscanf(fp, "%d,%d,%d,%d", &temp1, &temp2, &temp3, &temp4) == 4) {
        (*input1)[counter] = temp1;
        (*input2)[counter] = temp2;
        (*input3)[counter] = temp3;
        (*input4)[counter] = temp4;
        counter++;
    }

    fclose(fp);
    return len;
}

__device__ int numNextLevelNodes = 0;
__device__ int nextLevelNodesQueue[5000000];

__global__ void block_queuing_kernel(int numCurrLevelNodes, int* currLevelNodes, int* nodeNeighbors, int* nodePtrs, int* nodeVisited, int* nodeInput, int* nodeOutput_cuda, int* nodeGate, int queueSize){
    
    // Initialize shared memory queue
    extern __shared__ int sharedBlockQueue[];
    __shared__ int sharedBlockQueueSize, blockGlobalQueueIdx;

    if (threadIdx.x == 0)
        sharedBlockQueueSize = 0; 

    __syncthreads();

    int threadIndex = threadIdx.x + (blockDim.x * blockIdx.x);

    // Loop over all nodes in current level
    for (int id = threadIndex; id < numCurrLevelNodes; id++) {
        int nodeIdx = currLevelNodes[id];      

        // Loop over all neighbors of the node
        for (int nId = nodePtrs[nodeIdx]; nId < nodePtrs[nodeIdx+1]; nId++) {          
            int neighborIdx = nodeNeighbors[nId];

            // If the neighbor has not been visited yet
            const int visited = atomicExch(&(nodeVisited[neighborIdx]), 1);
            if (!(visited)) {
                const int queueIdx = atomicAdd(&sharedBlockQueueSize, 1);
                int result = 0;
                int nodeGateVal = nodeGate[neighborIdx];
                int nodeInputVal = nodeInput[neighborIdx];
                int nodeOutputVal = nodeOutput_cuda[nodeIdx];

                switch (nodeGateVal) {
                    case 0:
                        result = nodeInputVal & nodeOutputVal;
                        break;
                    case 1:
                        result = nodeInputVal | nodeOutputVal;
                        break;
                    case 2:
                        result = !(nodeInputVal & nodeOutputVal);
                        break;
                    case 3:
                        result = !(nodeInputVal | nodeOutputVal);
                        break;
                    case 4:
                        result = nodeInputVal ^ nodeOutputVal;
                        break;
                    case 5:
                        result = !(nodeInputVal ^ nodeOutputVal);
                        break;
                }
        
                // Update node output
                nodeOutput_cuda[neighborIdx] = result; 

                // Add to block queue if not full else add to global queue
                if (queueIdx < queueSize)
                    sharedBlockQueue[queueIdx] = neighborIdx;                  
                else {
                    sharedBlockQueueSize = queueSize;
                    const int GlIdx = atomicAdd(&numNextLevelNodes, 1);
                    nextLevelNodesQueue[GlIdx] = neighborIdx; 
                }
            }      
        }
    }
    
    __syncthreads();

    if (threadIdx.x == 0)
        blockGlobalQueueIdx = atomicAdd(&numNextLevelNodes, sharedBlockQueueSize);
  
    __syncthreads();

    // store block queue in global queue
    for (int i = threadIdx.x; i < sharedBlockQueueSize; i += blockDim.x)
        nextLevelNodesQueue[blockGlobalQueueIdx + i] = sharedBlockQueue[i];
}

int main(int argc, char *argv[]){

    // ~~~~~~~~~~~~~~~~~~~~~~~
    // step 1: parse arguments
    // ~~~~~~~~~~~~~~~~~~~~~~~
    
    if (argc < 10) {
        printf("Require parameters in the following order: <numBlock> <blockSize> <sharedQueueSize> <path_to_input_1.raw> <path_to_input_2.raw> <path_to_input_3.raw> <path_to_input_4.raw> <output_nodeOutput_filepath> <output_nextLevelNodes_filepath>.\n");
        exit(1);
    }

    int numNodePtrs;
    int numNodes;
    int *nodePtrs_h;
    int *nodeNeighbors_h;
    int *nodeVisited_h;
    int numTotalNeighbors_h;
    int *currLevelNodes_h;
    int numCurrLevelNodes;
    int numNextLevelNodes_h = 0;
    int *nodeGate_h;
    int *nodeInput_h;
    int *nodeOutput_h;

    const int blockSize = atoi(argv[1]);
    const int numBlocks = atoi(argv[2]);
    const int queueSize = atoi(argv[3]);

    char* input1 = argv[4];
    char* input2 = argv[5];
    char* input3 = argv[6];
    char* input4 = argv[7];

    char* nodeOutputFilename = argv[8];
    char* nextLevelNodesFilename = argv[9];

    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // step 2: read in inputs from file
    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

    numNodePtrs = read_input_one_two_four(&nodePtrs_h, input1);
    numTotalNeighbors_h = read_input_one_two_four(&nodeNeighbors_h, input2);
    numNodes = read_input_three(&nodeVisited_h, &nodeGate_h, &nodeInput_h, &nodeOutput_h, input3);
    numCurrLevelNodes = read_input_one_two_four(&currLevelNodes_h, input4);

    // ~~~~~~~~~~~~~~~~~~~~~~~~
    // step 3: allocate for GPU
    // ~~~~~~~~~~~~~~~~~~~~~~~~

    // outputs
    int *nextLevelNodes_h = (int *)malloc(numNodes*sizeof(int));
    int *nextLevelNodes_cuda = (int *)malloc(numNodes*sizeof(int));
    hipMalloc (&nextLevelNodes_cuda, numCurrLevelNodes * sizeof(int));
    hipMemcpy(nextLevelNodes_cuda, nextLevelNodes_h, numCurrLevelNodes * sizeof(int), hipMemcpyHostToDevice);
  
    // copy to device
    int numNodesSize = numNodes * sizeof(int);
    int* currLevelNodes_cuda = (int*)malloc(numCurrLevelNodes * sizeof(int)) ; 
    hipMalloc (&currLevelNodes_cuda, numCurrLevelNodes * sizeof(int));
    hipMemcpy(currLevelNodes_cuda, currLevelNodes_h, numCurrLevelNodes * sizeof(int), hipMemcpyHostToDevice);

    int* nodeNeighbors_cuda = (int*)malloc(numTotalNeighbors_h * sizeof(int)) ; 
    hipMalloc (&nodeNeighbors_cuda, numTotalNeighbors_h * sizeof(int));
    hipMemcpy(nodeNeighbors_cuda, nodeNeighbors_h, numTotalNeighbors_h * sizeof(int), hipMemcpyHostToDevice);

    int* nodePtrs_cuda = (int*)malloc(numNodePtrs * sizeof(int)) ; 
    hipMalloc (&nodePtrs_cuda, numNodePtrs * sizeof(int));
    hipMemcpy(nodePtrs_cuda, nodePtrs_h, numNodePtrs * sizeof(int), hipMemcpyHostToDevice);

    int* nodeVisited_cuda = (int*)malloc(numNodesSize) ; 
    hipMalloc (&nodeVisited_cuda, numNodesSize);
    hipMemcpy(nodeVisited_cuda, nodeVisited_h,numNodesSize, hipMemcpyHostToDevice);

    int* nodeInput_cuda = (int*)malloc(numNodesSize) ; 
    hipMalloc (&nodeInput_cuda, numNodesSize);
    hipMemcpy(nodeInput_cuda, nodeInput_h, numNodesSize, hipMemcpyHostToDevice);

    int* nodeOutput_cuda = (int*)malloc(numNodesSize) ; 
    hipMalloc (&nodeOutput_cuda, numNodesSize);
    hipMemcpy(nodeOutput_cuda, nodeOutput_h, numNodesSize, hipMemcpyHostToDevice);

    int* nodeGate_cuda = (int*)malloc(numNodesSize) ; 
    hipMalloc (&nodeGate_cuda, numNodesSize);
    hipMemcpy(nodeGate_cuda, nodeGate_h, numNodesSize, hipMemcpyHostToDevice);

    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // step 4: time parallel execution
    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

    hipEvent_t startGPU, stopGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);

    hipEventRecord(startGPU);

    block_queuing_kernel<<<numBlocks, blockSize, queueSize*sizeof(int)>>>(numCurrLevelNodes, currLevelNodes_cuda, nodeNeighbors_cuda, nodePtrs_cuda, nodeVisited_cuda, nodeInput_cuda, nodeOutput_cuda, nodeGate_cuda, queueSize);
    checkCudaErr(hipDeviceSynchronize(), "Syncronization");
    checkCudaErr(hipGetLastError(), "GPU");

    hipEventRecord(stopGPU);
    hipEventSynchronize(stopGPU);

    float timeGPU;
    hipEventElapsedTime(&timeGPU, startGPU, stopGPU);

    printf("Parallel Explicit: %.6f ms\n", timeGPU);

    hipEventDestroy(startGPU);
    hipEventDestroy(stopGPU);

    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // step 5: write to file and done!
    // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

    int* outputBuffer;
    outputBuffer = (int*)malloc(numNodesSize);
    checkCudaErr(hipMemcpy(outputBuffer, nodeOutput_cuda, numNodesSize, hipMemcpyDeviceToHost), "Copying");

    hipMemcpyFromSymbol(&numNextLevelNodes_h, HIP_SYMBOL(numNextLevelNodes), sizeof(int), 0, hipMemcpyDeviceToHost);
    checkCudaErr(hipMemcpyFromSymbol(nextLevelNodes_h,HIP_SYMBOL(nextLevelNodesQueue), numNextLevelNodes_h * sizeof(int), 0, hipMemcpyDeviceToHost), "Copying");

    // write nodeOutput
    FILE *nodeOutputFile = fopen(nodeOutputFilename, "w");
    int counter = 0;
    fprintf(nodeOutputFile,"%d\n",numNodes);

    while (counter < numNodes) {
        fprintf(nodeOutputFile,"%d\n",(outputBuffer[counter]));
        counter++;
    }

    fclose(nodeOutputFile);

    // write nextLevelNodes
    FILE *nextLevelOutputFile = fopen(nextLevelNodesFilename, "w");
    counter = 0;
    fprintf(nextLevelOutputFile,"%d\n",numNextLevelNodes_h);

    while (counter < numNextLevelNodes_h) {
        fprintf(nextLevelOutputFile,"%d\n",(nextLevelNodes_h[counter]));
        counter++;
    }

    fclose(nextLevelOutputFile);

    // ~~~~~~~~~~~~~~~~~~~~~
    // step 6: free at last!
    // ~~~~~~~~~~~~~~~~~~~~~

    free(outputBuffer);
    free(nextLevelNodes_h);
    hipFree(nextLevelNodes_cuda);
    hipFree(currLevelNodes_cuda);
    hipFree(nodeNeighbors_cuda);
    hipFree(nodePtrs_cuda);
    hipFree(nodeVisited_cuda);
    hipFree(nodeInput_cuda);
    hipFree(nodeOutput_cuda);
    hipFree(nodeGate_cuda);
}